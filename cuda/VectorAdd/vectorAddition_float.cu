
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cassert>
#include <iostream>
#include <chrono>

using namespace std;

__global__ void vectorAdd(float *a, float *b, float *c, int N){
    //Calculate thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    //Range check
    if(tid < N) c[tid] = a[tid] + b[tid];

}

void init_array(float *a, int N){
    for(int i=0; i<N; i++){
        a[i] = rand() % 100 + 0.234;
    }
}

void verifyArray(float *a, float *b, float *c, float N){
    for(int i=0; i<N; i++){
        assert(a[i] + b[i] == c[i]);
    }
}

int main(){
    int N = 1 << 25;
    size_t bytes = N * sizeof(bytes);

    float *a, *b, *c;

    //Allocate memory for arrays
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    //Initialize Data
    init_array(a, N);
    init_array(b, N);

    //CTA and Grid dimensions
    int THREADS = 1024;
    int BLOCKS = (int) (N + THREADS - 1)/THREADS;

    //Call Kernel
    std::chrono::steady_clock::time_point begin, end;
    begin = std::chrono::steady_clock::now();
    vectorAdd<<<BLOCKS, THREADS>>>(a, b, c, N);
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    std::cout << "GPU time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << " milliseconds" << std::endl;
    //Verify solution
    begin = std::chrono::steady_clock::now();
    verifyArray(a, b, c, N);
    end = std::chrono::steady_clock::now();
    std::cout << "CPU time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << " milliseconds" << std::endl;
    cout << "SUCCESSFUL" << endl;    

    return 0;
}