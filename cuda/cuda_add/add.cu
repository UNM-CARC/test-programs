#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}


// CPU version is the same - function to add the elements of two arrays
//void add(int n, float *x, float *y)
//{
//  for (int i = 0; i < n; i++)
//    y[i] = x[i] + y[i];
//}

int main(void)
{
  int N = 1<<20; // 1M elements

  // Allocate Unified Memory -- accessible from CPU or GPU
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));


  // Previous CPU Code
  //float *x = new float[N];
  //float *y = new float[N];

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);

  // Previous CPU code - Run kernel on 1M elements on the CPU
  //add(N, x, y);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  // Previous CPU Free memory
  // delete [] x;
  // delete [] y;

  return 0;
}
