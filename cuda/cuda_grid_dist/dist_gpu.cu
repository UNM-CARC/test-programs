
#include <hip/hip_runtime.h>
#include <math.h> //Include standard math library containing sqrt.
#include <chrono>
#include <iostream>
#include <stdio.h>
#include <sstream>
using namespace std::chrono;
using namespace std;

// Convenience function to format human friendly elapsed time 
#include <iomanip>
#include <optional>
#include <ostream>

template<typename T>
inline std::string format(T timeunit) {
  nanoseconds ns = duration_cast<nanoseconds>(timeunit);
  std::ostringstream os;
  bool foundNonZero  = false;
  os.fill('0');
  typedef duration<int, std::ratio<86400*365>> years;
  const auto y = duration_cast<years>(ns);
  if (y.count()) {
    foundNonZero = true;
    os << y.count() << "y:";
    ns -= y;
  }
  typedef duration<int, std::ratio<86400>> days;
  const auto d = duration_cast<days>(ns);
  if (d.count()) {
    foundNonZero = true;
    os << d.count() << "d:";
    ns -= d;
  }
  const auto h = duration_cast<hours>(ns);
  if (h.count() || foundNonZero) {
    foundNonZero = true;
    os << h.count() << "h:";
    ns -= h;
  }
  const auto m = duration_cast<minutes>(ns);
  if (m.count() || foundNonZero) {
    foundNonZero = true;
    os << m.count() << "m:";
    ns -= m;
  }
  const auto s = duration_cast<seconds>(ns);
  if (s.count() || foundNonZero) {
    foundNonZero = true;
    os << s.count() << "s:";
    ns -= s;
  }
  const auto ms = duration_cast<milliseconds>(ns);
  if (ms.count() || foundNonZero) {
    if (foundNonZero) {
      os << std::setw(3);
    }
    os << ms.count() << "ms.";
    ns -= ms;
    foundNonZero = true;
  }
  const auto us = duration_cast<microseconds>(ns);
  if (us.count() || foundNonZero) {
    if (foundNonZero) {
      os << std::setw(3);
    }
    os << us.count() << ".";
    ns -= us;
  }
  os << std::setw(3) << ns.count() << "ns" ;
  return os.str();
}

// A scaling function to convert integers 0,1,...,N-1 to evenly spaced floats 
//float scale(int i, int n)
//{
//  return ((float)i) / (n - 1);
//}

__device__ float scale( int i, int n )
{
  return ((float)i)/(n-i);
}

// Compute the distance between 2 points on a line.
//float distance(float x1, float x2)
//{
//  return sqrt((x2 - x1)*(x2 - x1));
//}

__device__ float distance( float x1, float x2 )
{
  return sqrt((x2 - x1)*(x2 - x1));
}



// Called on the CPU and RUNs on the GPU
__global__ void distanceKernel( float* device_out, float ref, int len)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const float x = scale(i, len);
  device_out[i] = distance(x, ref);
  //printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, device_out[i]);
}

// main function
int main( int argc, char* argv[])
{
  int N = atoi(argv[1]);
  // float out[N] = {0.0};

  
  // Choose a reference value from which distances are measured.
  const float ref = 0.5;

  auto t1 = high_resolution_clock::now();
  //for (int i = 0; i < N; ++i)
  //  {
  //    float x = scale(i, N);
  //    out[i] = distance(x, ref);
  //  }

  // Create the array to store data
  float* device_out = NULL;
  // Allocate the array on the GPU
  hipMalloc(&device_out, N*sizeof(float));

  // Replace the for loop with a CUDA block of computational threads
  // threads_per_block
  int threads_per_block = 32;
  int num_blocks = (N+threads_per_block-1)/threads_per_block;

  distanceKernel<<<num_blocks, threads_per_block>>>(device_out, ref, N);
  
  hipError_t err = hipDeviceSynchronize();
  cout << "GPU computation reports " << hipGetErrorString(err) << "." << endl;

  float* host_out = (float*) malloc(N*sizeof(float));
  hipMemcpy(host_out, device_out, N*sizeof(float), hipMemcpyDeviceToHost);  

  auto t2 = high_resolution_clock::now();

  /* Getting number of milliseconds as an integer. */
  auto duration_ms = duration_cast<milliseconds>(t2 - t1);

  cout.precision(3);

  cout << "Calcuated " << float(N) <<  " Distances in Time: " << format( duration_ms ) << endl;  
  cout << "First distance: " << host_out[0] << endl;
  cout << "Second distance: " << host_out[1] << endl;
  cout << "Last distance: " << host_out[N-1] << endl;
 
  hipFree(device_out);	  
  free(host_out);

  return 0;
}
