
#include <hip/hip_runtime.h>
#include <math.h> //Include standard math library containing sqrt.
#include <chrono>
#include <iostream>
#include <stdio.h>
#include <sstream>
using namespace std::chrono;

// Convenience function to format human friendly elapsed time
#include <iomanip>
#include <optional>
#include <ostream>
using namespace std;

__device__ float scale(int i, int n)
{
	return ((float)i)/(n - 1);
}
__device__ float distance(float x1, float x2)
{
	return sqrt((x2 - x1)*(x2 - x1));
}
__global__ void distanceKernel(float *device_out, float ref, int len)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = scale(i, len);
	device_out[i] = distance(x, ref);
	printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, device_out[i]);
}

int main(int argc, char* argv[])
{   
    int N = atoi(argv[1]);
    int	TPB = N;
    	const float ref = 0.5f;
	// Declare a pointer for an array of floats
	float *d_out = 0;
	// Allocate device memory to store the output array
	hipMalloc(&d_out, N*sizeof(float));
	// Launch kernel to compute and store distance values
	distanceKernel<<<N/TPB, TPB>>>(d_out, ref, N);

	float* host_out = (float*) malloc(N*sizeof(float));	
  	hipMemcpy(host_out, d_out, N*sizeof(float), hipMemcpyDeviceToHost);

	std::cout << host_out[N-1] << std::endl;

	hipFree(d_out); // Free the memory
	return 0;
}