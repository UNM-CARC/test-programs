
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cassert>
#include <iostream>
#include <chrono>

using namespace std;

__global__ void matrixMul(const float *a, const float *b, float *c, int N) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
  c[row * N + col] = 0;
  for (int k = 0; k < N; k++) {
    // Accumulate results for a single element
    c[row * N + col] += a[row * N + k] * b[k * N + col];
  }
}

void verify_result(float *a, float *b, float *c, int N){
  float temp;
  for(int i=0; i<N; i++){
    for(int j=0; j<N; j++){
      temp = 0;
      for(int k=0; k<N; k++){
        temp += a[i * N + k] * b[k * N + j];
      }
      assert(temp == c[i * N + j]);
    }
  }
}

//Initialize a square matrix
void init_matrix(float *m, int N){
  for(int i=0; i<N; i++){
    m[i] = rand() % 100 + 0.234;
  }
}

int main(){
  //Set square matrix dimension
  int N = 1 << 10; //1024
  size_t bytes = N * N * sizeof(float); //Declare size of matrix

  //Allocate memory for matrices
  float *a, *b, *c;
  hipMallocManaged(&a, bytes);
  hipMallocManaged(&b, bytes);
  hipMallocManaged(&c, bytes);

  // Initialize matrix
  init_matrix(a, N);
  init_matrix(b, N);

  //Set CTA and Grid dimensions
  int threads = 32; //Will be squared -> 256
  int blocks = (N + threads - 1)/threads;

  //Kernel launch parameters
  dim3 THREADS(threads, threads);
  dim3 BLOCKS(blocks, blocks);

  //Launch Kernel
  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
  matrixMul<<<BLOCKS, THREADS>>>(a, b, c, N);
  hipDeviceSynchronize();
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
  std::cout << "GPU time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << " milliseconds" << std::endl;
  //Verify results
  begin = std::chrono::steady_clock::now();
  verify_result(a, b, c, N);
  end = std::chrono::steady_clock::now();
  std::cout << "CPU time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << " milliseconds" << std::endl;
  cout << "SUCCESSFUL" << endl;
  return 0;
}